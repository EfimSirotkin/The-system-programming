
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <windows.h>
#include <iostream>
#include <fstream>
#include <string>

#define CONVERT 1024*1024;
#define PERCENTAGE 100.0

using namespace std;

size_t GetGraphicDeviceVRamUsage(int _NumGPU)
{
	hipSetDevice(_NumGPU);

	size_t l_free = 0;
	size_t l_Total = 0;
	hipError_t error_id = hipMemGetInfo(&l_free, &l_Total);
	double gpuUsage = l_free;
	gpuUsage /= l_Total;
	gpuUsage *= 100.0;
	cout << gpuUsage << endl;

	return (l_Total - l_free);
}
extern "C" __declspec(dllexport) double RetrieveGpuUsage() {
	hipSetDevice(0);
	size_t l_free = 0;
	size_t l_Total = 0;
	hipError_t error_id = hipMemGetInfo(&l_free, &l_Total);
	hipDeviceReset();
	return (double) PERCENTAGE * l_free/l_Total/ CONVERT;
}
extern "C" __declspec(dllexport) void printDiscreteGPUInfo() {
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %.3f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
		printf("  Clock Rate (KHz): %d\n",
			prop.clockRate);
		printf("  L2CacheSize (bytes): %d\n", prop.l2CacheSize);
		printf("  Global memory(mbytes): %d\n", (prop.totalGlobalMem/1024/1024));

	}
}
extern "C" __declspec(dllexport) void saveGpuInfoToFile(ofstream& on)
{
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		on << "==========   ���������� � ���������� ����������   ============" << endl;
		on << "  Device Number:  " << i << endl;
		on << "  Device name: " << prop.name << endl;
		on << "  Memory Clock Rate (KHz): " <<
			prop.memoryClockRate << endl;
		on << "  Memory Bus Width (bits): " <<
			prop.memoryBusWidth << endl;
		on << "  Peak Memory Bandwidth (GB/s): " <<
			(int)2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6;
		on << "  Clock Rate (KHz): " <<
			prop.clockRate << endl;
		on << "  L2CacheSize (bytes): " << prop.l2CacheSize << endl;
		on << "  Global memory(mbytes): " << prop.totalGlobalMem / 1024 / 1024 << endl;

	}
}
